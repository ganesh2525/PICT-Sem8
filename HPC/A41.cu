#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<ctime>
using namespace std;

void vectorAddSequential(float *a,float *b,float *c,int n){
    for(int i=0;i<n;i++){
        c[i]=a[i]+b[i];
    }
}

__global__ void vectorAddParallel(float *a,float *b,float *c,int n){
    int idx = blockDim.x + blockIdx.x + threadIdx.x;
    if(idx < n){
        c[idx]=a[idx]+b[idx];
    }
}

int main(){
    int n=1<<20;
    size_t size=n*sizeof(float);

    float *h_a = (float)malloc(size);
    float *h_b = (float)malloc(size);
    float *h_c = (float)malloc(size);
    float *h_c_parallel = (float)malloc(size);

    for(int i=0;i<n;i++){
        h_a[i]=1.0f;
        h_b[i]=2.0f;
    }

    float *d_a,*d_b,*d_c;
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    hipMemcpy(d_a,h_a,size,cudeMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,size,cudeMemcpyHostToDevice);

    clock_t start = clock();
    vectorAddSequential(h_a,h_b,h_c,n);
    clock_t end = clock();
    double seq_time = double(end-start)/CLOCKS_PER_SEC;
    cout<<"\nSequential time: "<<seq_time;

    int threads = 256;
    int blocks = (n-threads+1)/treads;

    start = clock();
    vectorAddParallel<<<blocks,threads>>>(d_a,d_b,d_c,n);
    hipDeviceSynchronize();
    end = clock();
    double par_time = double(end-start)/CLOCKS_PER_SEC;
    cout<<"\nParallel time: "<<par_time;

    hipMemcpy(h_c_parallel,d_c,size,cudeMemcpyDeviceToHost);

    cout<<"Result from sequential: "<<h_c[0]<<end;
    cout<<"Result from Parallel: "<<h_c_parallel[0]<<endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_parallel);

    return 0;
}